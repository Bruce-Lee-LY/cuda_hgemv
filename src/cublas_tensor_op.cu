// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:49:00 on Mon, Oct 09, 2023
//
// Description: cublas tensor op hgemv

#include "common.h"

hipblasHandle_t getCublasTensorOpHandle() {
    hipblasHandle_t handle = nullptr;
    HGEMV_CHECK_CUBLAS_ERROR(hipblasCreate(&handle));
    HGEMV_CHECK_CUBLAS_ERROR(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    return handle;
}

void cublasTensorOp(half *A, half *B, half *C, size_t N, size_t K) {
    static hipblasHandle_t handle = getCublasTensorOpHandle();
    static size_t M = 1;
    static float alpha = 1.0;
    static float beta = 0.0;

    HGEMV_CHECK_CUBLAS_ERROR(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A,
                                          HIP_R_16F, K, &beta, C, HIP_R_16F, N, HIPBLAS_COMPUTE_32F,
                                          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}
