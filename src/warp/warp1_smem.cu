#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:49:00 on Mon, Oct 09, 2023
//
// Description: warp1 smem hgemv

#include "common.h"

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 4
#define THREADS_PER_BLOCK 128  // WARP_SIZE * WARPS_PER_BLOCK

__global__ void warp1SmemKernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, size_t N,
                                size_t K) {
    extern __shared__ half A_smem[];
    size_t A_smem_iters = div_ceil(K, THREADS_PER_BLOCK);
#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        size_t idx = i * THREADS_PER_BLOCK + threadIdx.x;
        A_smem[idx] = A[idx];
    }

    __syncthreads();

    const size_t warp_id = threadIdx.x / WARP_SIZE;
    const size_t warp_col = blockIdx.x * WARPS_PER_BLOCK + warp_id;
    if (warp_col >= N) {
        return;
    }

    const size_t K_iters = div_ceil(K, WARP_SIZE);
    const size_t lane_id = threadIdx.x % WARP_SIZE;

    float tmp = 0.0;
#pragma unroll
    for (size_t i = 0; i < K_iters; ++i) {
        size_t A_idx = i * WARP_SIZE + lane_id;
        size_t B_idx = i * WARP_SIZE + lane_id + warp_col * K;
        tmp += __half2float(A_smem[A_idx]) * __half2float(B[B_idx]);
    }

    constexpr unsigned int mask = 0xffffffff;
#pragma unroll
    for (size_t i = WARP_SIZE / 2; i >= 1; i /= 2) {
        tmp += __shfl_xor_sync(mask, tmp, i);
    }

    if (lane_id == 0) {
        C[warp_col] = __float2half(tmp);
    }
}

size_t initWarp1Smem(size_t K) {
    int dev_id = 0;
    HGEMV_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMV_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size = K * sizeof(half);
    HLOG("smem_max_size: %.0f KBytes (%zu bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMV_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMV_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(warp1SmemKernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void warp1Smem(half *A, half *B, half *C, size_t N, size_t K) {
    static size_t smem_max_size = initWarp1Smem(K);

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(div_ceil(N, WARPS_PER_BLOCK));

    warp1SmemKernel<<<grid, block, smem_max_size>>>(A, B, C, N, K);
}
