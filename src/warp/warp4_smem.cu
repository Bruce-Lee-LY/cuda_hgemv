#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:49:00 on Mon, Oct 09, 2023
//
// Description: warp4 smem hgemv

#include "common.h"

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 4
#define THREADS_PER_BLOCK 128  // WARP_SIZE * WARPS_PER_BLOCK

#define COLS_PER_WARP 4
#define COLS_PER_BLOCK 16    // COLS_PER_WARP * WARPS_PER_BLOCK
#define THREADS_PER_GROUP 8  // WARP_SIZE / COLS_PER_WARP

__global__ void warp4SmemKernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, size_t N,
                                size_t K) {
    extern __shared__ half A_smem[];
    size_t A_smem_iters = div_ceil(K, THREADS_PER_BLOCK);
#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        size_t idx = i * THREADS_PER_BLOCK + threadIdx.x;
        A_smem[idx] = A[idx];
    }

    __syncthreads();

    const size_t group_id = threadIdx.x / THREADS_PER_GROUP;
    const size_t group_col = blockIdx.x * COLS_PER_BLOCK + group_id;
    if (group_col >= N) {
        return;
    }

    const size_t K_iters = div_ceil(K, THREADS_PER_GROUP);
    const size_t group_lane_id = threadIdx.x % THREADS_PER_GROUP;

    float tmp = 0.0;
#pragma unroll
    for (size_t i = 0; i < K_iters; ++i) {
        size_t A_idx = i * THREADS_PER_GROUP + group_lane_id;
        size_t B_idx = i * THREADS_PER_GROUP + group_lane_id + group_col * K;
        tmp += __half2float(A_smem[A_idx]) * __half2float(B[B_idx]);
    }

    constexpr unsigned int mask = 0xffffffff;
#pragma unroll
    for (size_t i = THREADS_PER_GROUP / 2; i >= 1; i /= 2) {
        tmp += __shfl_xor_sync(mask, tmp, i);
    }

    if (group_lane_id == 0) {
        C[group_col] = __float2half(tmp);
    }
}

size_t initWarp4Smem(size_t K) {
    int dev_id = 0;
    HGEMV_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMV_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size = K * sizeof(half);
    HLOG("smem_max_size: %.0f KBytes (%zu bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMV_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMV_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(warp4SmemKernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void warp4Smem(half *A, half *B, half *C, size_t N, size_t K) {
    static size_t smem_max_size = initWarp4Smem(K);

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(div_ceil(N, COLS_PER_BLOCK));

    warp4SmemKernel<<<grid, block, smem_max_size>>>(A, B, C, N, K);
}
