#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:49:00 on Mon, Oct 09, 2023
//
// Description: hgemv main

#include "gflags/gflags.h"
#include "omp.h"
#include "tester.h"

#define HGEMV_FUNC(name) void name(half *A, half *B, half *C, size_t N, size_t K)

HGEMV_FUNC(cublasTensorOp);

HGEMV_FUNC(threadNaive);
HGEMV_FUNC(threadSmem);

HGEMV_FUNC(warp1Naive);
HGEMV_FUNC(warp1Smem);
HGEMV_FUNC(warp2Naive);
HGEMV_FUNC(warp2Smem);
HGEMV_FUNC(warp4Naive);
HGEMV_FUNC(warp4Smem);
HGEMV_FUNC(warp8Naive);
HGEMV_FUNC(warp8Smem);
HGEMV_FUNC(warp16Naive);
HGEMV_FUNC(warp16Smem);

DEFINE_uint32(N, 256, "N");
DEFINE_uint32(K, 128, "K");
DEFINE_uint32(warmup_iterations, 1, "warmup iteration numbers and average the result");
DEFINE_uint32(profiling_iterations, 10, "profiling iteration numbers and average the result");
DEFINE_uint32(sleep_duration, 100, "sleep_milliseconds between profiling");
DEFINE_bool(enable_check, false, "check the GPU result against the cublas result");
DEFINE_uint32(cpu_procs, omp_get_num_procs(), "processor num used of CPU");
DEFINE_uint32(gpu_rank, 0, "the used GPU rank");

int main(int argc, char *argv[]) {
    GFLAGS_NAMESPACE::ParseCommandLineFlags(&argc, &argv, true);

    omp_set_num_threads(FLAGS_cpu_procs);
    HGEMV_CHECK_CUDART_ERROR(hipSetDevice(FLAGS_gpu_rank));

    hipDeviceProp_t dev_prop;
    HGEMV_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, FLAGS_gpu_rank));
    HLOG("CUDA HGEMV start with %u CPU processes on the %u-th GPU: %s", FLAGS_cpu_procs, FLAGS_gpu_rank, dev_prop.name);

    int driver_version = 0;
    int runtime_version = 0;
    HGEMV_CHECK_CUDART_ERROR(hipDriverGetVersion(&driver_version));
    HGEMV_CHECK_CUDART_ERROR(hipRuntimeGetVersion(&runtime_version));
    HLOG("CUDA driver version / runtime version: %d.%d / %d.%d", driver_version / 1000, (driver_version % 100) / 10,
         runtime_version / 1000, (runtime_version % 100) / 10);
    HLOG("CUDA capability major/minor version number: %d.%d", dev_prop.major, dev_prop.minor);
    HLOG("%d multiprocessors, %d CUDA cores/MP: %d CUDA cores", dev_prop.multiProcessorCount,
         convert_SM_to_cores(dev_prop.major, dev_prop.minor),
         convert_SM_to_cores(dev_prop.major, dev_prop.minor) * dev_prop.multiProcessorCount);
    HLOG("GPU max clock rate: %.0f MHz (%0.2f GHz)", static_cast<double>(dev_prop.clockRate) * 1e-3,
         static_cast<double>(dev_prop.clockRate) * 1e-6);
    HLOG("Memory clock rate: %.0f MHz (%0.2f GHz)", static_cast<double>(dev_prop.memoryClockRate) * 1e-3,
         static_cast<double>(dev_prop.memoryClockRate) * 1e-6);
    HLOG("Memory bus width: %d-bit", dev_prop.memoryBusWidth);
    HLOG("Total amount of global memory: %.0f MBytes (%zu Bytes)",
         static_cast<double>(dev_prop.totalGlobalMem) / 1048576, dev_prop.totalGlobalMem);
    HLOG("Total amount of constant memory: %.0f KBytes (%zu Bytes)", static_cast<double>(dev_prop.totalConstMem) / 1024,
         dev_prop.totalConstMem);
    HLOG("Total amount of shared memory per block: %.0f KBytes (%zu Bytes)",
         static_cast<double>(dev_prop.sharedMemPerBlock) / 1024, dev_prop.sharedMemPerBlock);
    HLOG("Total shared memory per multiprocessor: %.0f KBytes (%zu Bytes)",
         static_cast<double>(dev_prop.sharedMemPerMultiprocessor) / 1024, dev_prop.sharedMemPerMultiprocessor);
    HLOG("L2 cache size: %.0f KBytes (%d Bytes)", static_cast<double>(dev_prop.l2CacheSize) / 1024,
         dev_prop.l2CacheSize);
    HLOG("Total number of registers available per block: %d", dev_prop.regsPerBlock);
    HLOG("Warp size: %d", dev_prop.warpSize);
    HLOG("Max number of threads per multiprocessor: %d", dev_prop.maxThreadsPerMultiProcessor);
    HLOG("Max number of threads per block: %d", dev_prop.maxThreadsPerBlock);
    HLOG("Max dimension size of a thread block (x,y,z): (%d, %d, %d)", dev_prop.maxThreadsDim[0],
         dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
    HLOG("Max dimension size of a grid size (x,y,z): (%d, %d, %d)", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1],
         dev_prop.maxGridSize[2]);

    HLOG("A (1 x %u) * B (%u x %u) = C (1 x %u)", FLAGS_K, FLAGS_K, FLAGS_N, FLAGS_N);
    HLOG("Profiling: warmup iterations: %u, profiling iterations: %u, sleep duration: %u ms, enable check: %d",
         FLAGS_warmup_iterations, FLAGS_profiling_iterations, FLAGS_sleep_duration, FLAGS_enable_check);

    Tester tester(FLAGS_N, FLAGS_K, FLAGS_warmup_iterations, FLAGS_profiling_iterations, FLAGS_sleep_duration,
                  FLAGS_enable_check);
    tester.evaluate(cublasTensorOp, "Cublas-Tensor-Op");

    tester.evaluate(threadNaive, "Thread-Naive");
    tester.evaluate(threadSmem, "Thread-Smem");

    tester.evaluate(warp1Naive, "Warp1-Naive");
    tester.evaluate(warp1Smem, "Warp1-Smem");
    tester.evaluate(warp2Naive, "Warp2-Naive");
    tester.evaluate(warp2Smem, "Warp2-Smem");
    tester.evaluate(warp4Naive, "Warp4-Naive");
    tester.evaluate(warp4Smem, "Warp4-Smem");
    tester.evaluate(warp8Naive, "Warp8-Naive");
    tester.evaluate(warp8Smem, "Warp8-Smem");
    tester.evaluate(warp16Naive, "Warp16-Naive");
    tester.evaluate(warp16Smem, "Warp16-Smem");

    GFLAGS_NAMESPACE::ShutDownCommandLineFlags();

    HLOG("Done");

    return 0;
}
